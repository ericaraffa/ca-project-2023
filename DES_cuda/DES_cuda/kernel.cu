#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <thread>

#include <fstream>
#include "matrices.h"
#include "numberConversions.h"

#include <vector>
#include <stdio.h>
#include <cstdio>
#include <iostream>
#include <string>
#include <chrono>

using namespace std;

//#define N_THREAD 16
#define N_BLOCKS 16
#define THREADS_PER_BLOCK 128
#define DES_BLOCK_SIZE 8
#define ROUNDKEY_SIZE 48
#define ROUNDKEY_NUM 16


void my_strncpy_h(char* dest, const char* src, int dim) {

    for (int i = 0; i < dim; i++) {
        dest[i] = src[i];
    }
}

void my_reverse_h(char* roundKeys, int len_key, int num_key) {

    int dim = len_key*num_key;
    
    char* end_ptr = roundKeys + (num_key-1) * len_key; //Last key
    char* begin_ptr = roundKeys; 
    char* tmp = (char*)malloc(len_key);

    /*
    printf("BEGIN\n");
    for (int j = 0; j < len_key; j++) { 
        printf("%c", *(begin_ptr + j)); 
    }
    printf("\n");

    printf("END\n");
    for (int j = 0; j < len_key; j++) { 
        printf("%c", *(end_ptr + j)); 
    }
    printf("\n");
    */
    

    for (int i = 0; i < (num_key)/2; i++) {

        my_strncpy_h(tmp, begin_ptr, len_key);
        my_strncpy_h(begin_ptr, end_ptr, len_key);
        my_strncpy_h(end_ptr, tmp, len_key); //old begin

        // update pointers positions
        begin_ptr = begin_ptr + len_key;
        end_ptr = end_ptr - len_key;
    }

    free(tmp);
}

__device__ void opt_permutation(char *key, int* arr, int n, char *ret)
{
    for (int i = 0; i < n; i++) {
        ret[i] = key[arr[i] - 1]; 
    }
}

__device__ void shift_left(char *k, int shifts, char *ret)
{
    for (int i = 0; i < 28; i++) {
        ret[i] = k[(i + shifts) % 28];
    }     
}

__device__ void opt_xorOperation(char* str1, char* str2, int n, char* xored)
{

    for (int i = 0; i < n; i++) {
        xored[i] = ((str1[i] - '0') ^ (str2[i] - '0')) + '0';
    }
}

__global__ void encryption(char* pt, char* roundKeys, char* ret_cipher, int pt_size, int groups_per_thread)
{

    extern __shared__ char roundKeys_s[];

    // Control
    long index_th = threadIdx.x * groups_per_thread * DES_BLOCK_SIZE;
    long index_block = blockIdx.x * blockDim.x * groups_per_thread * DES_BLOCK_SIZE;
    long index = index_th + index_block;

    if (index_th == 0) {
        my_strncpy(roundKeys_s, roundKeys, ROUNDKEY_SIZE * ROUNDKEY_NUM * sizeof(char));
    }
    __syncthreads();

    if (index < (pt_size - (groups_per_thread * DES_BLOCK_SIZE))) {

        // String to binary 
        char* pt_bin = (char*)malloc(DES_BLOCK_SIZE * 8);

        // For first permutation
        char* pt_perm = (char*)malloc(64);

        // For splitting 64 bits plain text to LPT and RPT of 32 bits each
        char* left = (char*)malloc(32);
        char* right = (char*)malloc(32);

        // For single roundKey
        char* expandedRPT = (char*)malloc(ROUNDKEY_SIZE);
        char* xored = (char*)malloc(ROUNDKEY_SIZE);

        //s_box_i string array for storing the 4 bits outputs 
        char* s_box_i = (char*)malloc(sizeof(int));

        //s_box_final stores the final result from s box. i.e. concat all the result array elements
        char* s_box_final = (char*)malloc(32);

        // For P-box permutation and xor
        char* p_box_perm = (char*)malloc(32);
        char* p_box_xored = (char*)malloc(32);

        // For final result
        char* cipher_perm = (char*)malloc(64);
        char* cipher = (char*)malloc(DES_BLOCK_SIZE);

        for (int i = 0; i < groups_per_thread; i++) {

            // String to binary 
            str2bin(&pt[index + (i * DES_BLOCK_SIZE)], DES_BLOCK_SIZE, pt_bin);

            // Initial permutation Process
            opt_permutation(pt_bin, initialPermutation, 64, pt_perm);

            // Splitting of 64bits plain text to LPT and RPT of 32 bits each
            my_strncpy(left, pt_perm, 32);
            my_strncpy(right, pt_perm + 32, 32);

            for (int j = 0; j < ROUNDKEY_NUM; j++) {
                // Expansion Permutation 
                opt_permutation(right, dBox, ROUNDKEY_SIZE, expandedRPT);

                // XOR of RoundKey and expandedRPT
                opt_xorOperation(roundKeys_s + (j * ROUNDKEY_SIZE), expandedRPT, ROUNDKEY_SIZE, xored);

                // S-boxes
                for (int k = 0; k < 8; k++) {
                    //the value of '0' is 48, '1' is 49 and so on. but since we are referring the matrix index, we are interested in 0,1,..
                    //So, the '0' should be subtracted . i.e. the 49 value of '1' will be 49-48=1.
                    int row = 2 * int(xored[k * 6]) + int(xored[k * 6 + 5]);
                    int col = 8 * int(xored[k * 6 + 1]) + 4 * int(xored[k * 6 + 2]) + 2 * int(xored[k * 6 + 3]) + int(xored[k * 6 + 4]);

                    int val = sbox[k][row][col];

                    decimalToBinary(val, s_box_i);

                    my_strncpy(s_box_final + (k * sizeof(int)), s_box_i, sizeof(int));
                }

                // P-Box Permutation 
                opt_permutation(s_box_final, pbox, 32, p_box_perm);
                // XOR of left and p_box_perm 
                opt_xorOperation(p_box_perm, left, 32, p_box_xored);
                // Update left
                my_strncpy(left, p_box_xored, 32);
                // Swap left and right in every rounds except the last round
                if (j != 15) {
                    char* tmp;
                    tmp = left;
                    left = right;
                    right = tmp;
                }
            }
            // Left and Right combined
            my_strncpy(pt_perm, left, 32);
            my_strncpy(pt_perm + 32, right, 32);

            // Final Permutation to obtain 64bits cipher text
            opt_permutation(pt_perm, finalPermutation, 64, cipher_perm);

            bin2str(cipher_perm, DES_BLOCK_SIZE * 8, cipher);      

            my_strncpy(&ret_cipher[index + (i * DES_BLOCK_SIZE)], cipher, DES_BLOCK_SIZE);

        }
        //FREE
        free(pt_bin);
        free(pt_perm);
        free(left);
        free(right);
        free(expandedRPT);
        free(xored);
        free(s_box_i);
        free(s_box_final);
        free(p_box_perm);
        free(p_box_xored);
        free(cipher_perm);
        free(cipher);
    }   
}

__global__ void generate_roundKeys(char *key, int key_size, char *ret_roundKeys, int roundKey_num) {

    char * key_bin = (char*)malloc(key_size * 4);
    char * key_perm_shift = (char*)malloc(56);

    convertToBinary(key, key_size, key_bin); 
    opt_permutation(key_bin, keyTransformation, 56, key_perm_shift); // key without parity 

    free(key_bin);
   
    //Splitting 56 bit keys to left and right of 28 bits each
    char *left = (char*)malloc(28);
    char *right = (char*)malloc(28);

    for (int i = 0; i < roundKey_num; i++) {
        //Left Shift and Right Shift done to the respective left and right keys in each round
        shift_left(key_perm_shift, shiftsMatrix[i], left);
        shift_left((key_perm_shift + 28), shiftsMatrix[i], right);
        my_strncpy(key_perm_shift, left, 28);
        my_strncpy(key_perm_shift + 28, right, 28);

        // Key Compression : Converting 56 bit key to 48 bit combined key
        opt_permutation(key_perm_shift, keyCompression, 48, (ret_roundKeys + (i * 48)));
    }

    free(key_perm_shift);
    free(left);
    free(right);
}


int main()
{

    hipEvent_t cuda_start, cuda_stop;  
    hipEventCreate(&cuda_start); 
    hipEventCreate(&cuda_stop); 
    float milliseconds_enc = 0; 
    float milliseconds_dec = 0;

    FILE* file = fopen("pt_10000kb.txt", "rb");
    if (!file) {
        cout << "dead" << endl;
    }

    char *pt;
    long FILE_SIZE = 0;
    long PT_SIZE = 0;
    long PADDING = 0;
    int c, n_groups; 

    fseek(file, 0L, SEEK_END);
    FILE_SIZE = ftell(file);
    rewind(file);

    while ((FILE_SIZE + PADDING) % DES_BLOCK_SIZE != 0) {
        PADDING++; 
    }

    n_groups = (FILE_SIZE + PADDING) / DES_BLOCK_SIZE;

    //Allocate mem for pt
    PT_SIZE = n_groups * DES_BLOCK_SIZE * sizeof(char);
    pt = (char*)malloc(PT_SIZE);

    //Copy the file
    for (int i = 0; i < n_groups; i++) {
        if (i == n_groups - 1) { 
            fread(pt + (i * DES_BLOCK_SIZE), DES_BLOCK_SIZE - PADDING, 1, file);

            for (int j = (i * DES_BLOCK_SIZE) + DES_BLOCK_SIZE - PADDING; j < n_groups * DES_BLOCK_SIZE; j++) {
                pt[j] = ' ';
            }
        }
        else {
            fread(pt + (i * DES_BLOCK_SIZE), DES_BLOCK_SIZE, 1, file);
        }
    }
    fclose(file);
    
    // GENERATE KEYS
    const int key_size = 17;
    char key[key_size] = "ABC12532110EDA56";
    char* roundKeys;

    roundKeys = (char*)malloc(ROUNDKEY_SIZE * ROUNDKEY_NUM * sizeof(char));


    // GPU
    char* d_roundKeys;
    char* d_key;

    hipMalloc((void**)&d_key, key_size); 
    hipMalloc((void**)&d_roundKeys, ROUNDKEY_SIZE * ROUNDKEY_NUM * sizeof(char));

    hipMemcpy(d_key, &key, key_size, hipMemcpyHostToDevice);

    generate_roundKeys<<<1,1>>>(d_key, key_size, d_roundKeys, ROUNDKEY_NUM);
    hipDeviceSynchronize();

    hipFree(d_key);

    hipMemcpy(roundKeys, d_roundKeys, ROUNDKEY_SIZE * ROUNDKEY_NUM *sizeof(char), hipMemcpyDeviceToHost);

    //	ENCRYPTION: creation of the vector cipher (of size pt.size()) and a vector of threads. After defining the N_THREAD the 'blocks'
    //				of the plain text are splitted by the num of threads.
    //				The function thread t() create a thread and it will be pushed inside the array threads.


    char* cipher = (char*)malloc(PT_SIZE);

    // GPU
    char* d_pt;
    char* d_cipher;

    // Allocate
    hipMalloc((void**)&d_pt, PT_SIZE);
    hipMalloc((void**)&d_cipher, PT_SIZE);
    // Copy inputs
    hipMemcpy(d_pt, pt, PT_SIZE, hipMemcpyHostToDevice);

    dim3 grid(N_BLOCKS);
    dim3 block(THREADS_PER_BLOCK);

    int groups_per_thread = n_groups / (block.x * grid.x);

    int shared_mem_dim = ROUNDKEY_SIZE * ROUNDKEY_NUM * sizeof(char);

    hipEventRecord(cuda_start); 
    encryption <<<grid, block, shared_mem_dim>>> (d_pt, d_roundKeys, d_cipher, PT_SIZE, groups_per_thread);
    hipEventRecord(cuda_stop);
 
    // Copy results
    hipMemcpy(cipher, d_cipher, PT_SIZE, hipMemcpyDeviceToHost);

    
    hipEventSynchronize(cuda_stop); 
    hipEventElapsedTime(&milliseconds_enc, cuda_start, cuda_stop); 
    
    // Decryption : Reversing the round keys and executing the encryption process to get Plain Text

    my_reverse_h(roundKeys, ROUNDKEY_SIZE, ROUNDKEY_NUM); //ok

    // GPU decrypt
    // Copy input
    hipMemcpy(d_roundKeys, roundKeys, ROUNDKEY_SIZE * ROUNDKEY_NUM * sizeof(char), hipMemcpyHostToDevice); 
    hipMemcpy(d_cipher, cipher, PT_SIZE, hipMemcpyHostToDevice); 

    hipEventRecord(cuda_start);
    encryption <<<grid, block, shared_mem_dim>>> (d_cipher, d_roundKeys, d_pt, PT_SIZE, groups_per_thread);
    hipEventRecord(cuda_stop);

    // Copy results
    hipMemcpy(pt, d_pt, PT_SIZE, hipMemcpyDeviceToHost); 
    
    hipEventSynchronize(cuda_stop);
    hipEventElapsedTime(&milliseconds_dec, cuda_start, cuda_stop);   

    // Cleanup
    hipFree(d_pt); 
    hipFree(d_roundKeys); 
    hipFree(d_cipher);
    
    free(pt);
    free(roundKeys);
    free(cipher);

    cout << "Time taken (encryption+decryption): " << milliseconds_enc + milliseconds_dec << " ms" << endl;

}
